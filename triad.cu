#include "hip/hip_runtime.h"
#include <functional>
#include <iostream>
#include <string>
#include <vector>

#include <sys/wait.h>
#include <unistd.h>

#include "cxxopts.hpp"

#include "cache.hpp"
#include "check_cuda.cuh"

struct Result {
  double kernel;
  double copy;
  double total;
};

template <typename T>
__global__ void triad_kernel(T *__restrict__ a, const T *__restrict__ b,
                             const T *__restrict__ c, const T scalar,
                             const size_t n) {
  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    a[i] = b[i] + scalar * c[i];
  }
}



// write a single int
__global__ void write_int(int *a) {
  if (0 == blockDim.x * blockIdx.x + threadIdx.x) {
    *a = 700;
  }
}

bool try_system_allocator() {
  int *a = new int;
  write_int<<<1,1>>>(a);
  hipError_t err = hipDeviceSynchronize();
  delete a;
  if (err == hipErrorIllegalAddress) {
    fprintf(stderr, "got illegal address using system allocator\n");
    return false;
  }
  CUDA_RUNTIME(hipDeviceReset());
  return true;
}

typedef enum {
  PAGEABLE,
  PINNED,
  ZERO_COPY,
  MANAGED,
  SYSTEM,
} AllocationType;

typedef enum {
  NONE = 0x0,
  ACCESS = 0x1,
  PREFETCH = 0x2,
} Hint;

inline Hint operator|(Hint a, Hint b) {
  return static_cast<Hint>(static_cast<int>(a) | static_cast<int>(b));
}

template <typename T>
Result benchmark_naive(size_t n, AllocationType at, Hint hint) {

  T *a_h = nullptr;
  T *b_h = nullptr;
  T *c_h = nullptr;

  switch (at) {
  case PAGEABLE:
  case SYSTEM:
    a_h = new T[n];
    b_h = new T[n];
    c_h = new T[n];
    break;
  case PINNED:
    CUDA_RUNTIME(hipHostAlloc(&a_h, n * sizeof(T), 0));
    CUDA_RUNTIME(hipHostAlloc(&b_h, n * sizeof(T), 0));
    CUDA_RUNTIME(hipHostAlloc(&c_h, n * sizeof(T), 0));
    break;
  case ZERO_COPY:
    CUDA_RUNTIME(hipHostAlloc(&a_h, n * sizeof(T), hipHostMallocMapped));
    CUDA_RUNTIME(hipHostAlloc(&b_h, n * sizeof(T), hipHostMallocMapped));
    CUDA_RUNTIME(hipHostAlloc(&c_h, n * sizeof(T), hipHostMallocMapped));
    break;
  case MANAGED:
    CUDA_RUNTIME(hipMallocManaged(&a_h, n * sizeof(T)));
    CUDA_RUNTIME(hipMallocManaged(&b_h, n * sizeof(T)));
    CUDA_RUNTIME(hipMallocManaged(&c_h, n * sizeof(T)));
    break;
  default:
    fprintf(stderr, "unexpected AllocationType\n");
    exit(1);
  }

  // touch all pages
  // fprintf(stderr, "touch all pages\n");
  for (size_t i = 0; i < n; i += 32) {
    a_h[i] = i;
    b_h[i] = i;
    c_h[i] = i;
  }

  // fprintf(stderr, "init dev pointers\n");
  T *a_d = nullptr;
  T *b_d = nullptr;
  T *c_d = nullptr;

  switch (at) {
  case PAGEABLE:
  case PINNED:
    CUDA_RUNTIME(hipMalloc(&a_d, sizeof(T) * n));
    CUDA_RUNTIME(hipMalloc(&b_d, sizeof(T) * n));
    CUDA_RUNTIME(hipMalloc(&c_d, sizeof(T) * n));
    break;
  case ZERO_COPY:
    CUDA_RUNTIME(hipHostGetDevicePointer(&a_d, a_h, 0));
    CUDA_RUNTIME(hipHostGetDevicePointer(&b_d, c_h, 0));
    CUDA_RUNTIME(hipHostGetDevicePointer(&c_d, c_h, 0));
    break;
  case MANAGED:
  case SYSTEM:
    a_d = a_h;
    b_d = b_h;
    c_d = c_h;
    break;
  }

  // fprintf(stderr, "create events\n");
  hipEvent_t kernelStart, kernelStop;
  hipEvent_t txStart, txStop;
  hipEvent_t rxStart, rxStop;
  CUDA_RUNTIME(hipEventCreate(&kernelStart));
  CUDA_RUNTIME(hipEventCreate(&kernelStop));
  CUDA_RUNTIME(hipEventCreate(&txStart));
  CUDA_RUNTIME(hipEventCreate(&txStop));
  CUDA_RUNTIME(hipEventCreate(&rxStart));
  CUDA_RUNTIME(hipEventCreate(&rxStop));

  // flush caches
  // fprintf(stderr, "flush\n");
  flush_all(a_h, sizeof(T) * n);
  flush_all(b_h, sizeof(T) * n);
  flush_all(c_h, sizeof(T) * n);

  // fprintf(stderr, "h2d\n");
  CUDA_RUNTIME(hipEventRecord(txStart));
  if (at == PAGEABLE || at == PINNED) {
    CUDA_RUNTIME(hipMemcpyAsync(a_d, a_h, sizeof(T) * n, hipMemcpyDefault));
    CUDA_RUNTIME(hipMemcpyAsync(b_d, b_h, sizeof(T) * n, hipMemcpyDefault));
    CUDA_RUNTIME(hipMemcpyAsync(c_d, c_h, sizeof(T) * n, hipMemcpyDefault));
  }
  if ((at == MANAGED) && (PREFETCH & hint)) {
    CUDA_RUNTIME(hipMemPrefetchAsync(a_d, sizeof(T) * n, 0));
    CUDA_RUNTIME(hipMemPrefetchAsync(b_d, sizeof(T) * n, 0));
    CUDA_RUNTIME(hipMemPrefetchAsync(c_d, sizeof(T) * n, 0));
  }
  if ((at == MANAGED) && (ACCESS & hint)) {
    CUDA_RUNTIME(
        hipMemAdvise(a_d, sizeof(T) * n, hipMemAdviseSetAccessedBy, 0));
    CUDA_RUNTIME(
        hipMemAdvise(b_d, sizeof(T) * n, hipMemAdviseSetAccessedBy, 0));
    CUDA_RUNTIME(
        hipMemAdvise(c_d, sizeof(T) * n, hipMemAdviseSetAccessedBy, 0));
  }
  CUDA_RUNTIME(hipEventRecord(txStop));

  int dimBlock = 512;
  int dimGrid = (n + dimBlock - 1) / dimBlock;

  // fprintf(stderr, "launch\n");
  CUDA_RUNTIME(hipEventRecord(kernelStart));
  triad_kernel<<<dimGrid, dimBlock>>>(a_d, b_d, c_d, 1, n);
  CUDA_RUNTIME(hipEventRecord(kernelStop));

  // fprintf(stderr, "d2h\n");
  CUDA_RUNTIME(hipEventRecord(rxStart));
  if (at == PAGEABLE || at == PINNED) {
    CUDA_RUNTIME(hipMemcpyAsync(c_h, c_d, sizeof(T) * n, hipMemcpyDefault));
  }
  CUDA_RUNTIME(hipEventRecord(rxStop));

  // fprintf(stderr, "times\n");
  CUDA_RUNTIME(hipDeviceSynchronize());
  float txMillis;
  CUDA_RUNTIME(hipEventElapsedTime(&txMillis, txStart, txStop));
  float rxMillis;
  CUDA_RUNTIME(hipEventElapsedTime(&rxMillis, rxStart, rxStop));
  float kernelMillis;
  CUDA_RUNTIME(hipEventElapsedTime(&kernelMillis, kernelStart, kernelStop));
  float totalMillis;
  CUDA_RUNTIME(hipEventElapsedTime(&totalMillis, txStart, rxStop));

  // fprintf(stderr, "cuda free\n");
  if (at == PAGEABLE || at == PINNED) {
    CUDA_RUNTIME(hipFree(a_d));
    CUDA_RUNTIME(hipFree(b_d));
    CUDA_RUNTIME(hipFree(c_d));
  }

  // fprintf(stderr, "host free\n");
  switch (at) {
  case PAGEABLE:
  case SYSTEM:
    delete[] a_h;
    delete[] b_h;
    delete[] c_h;
    break;
  case PINNED:
  case ZERO_COPY:
    CUDA_RUNTIME(hipHostFree(a_h));
    CUDA_RUNTIME(hipHostFree(b_h));
    CUDA_RUNTIME(hipHostFree(c_h));
    break;
  case MANAGED:
    CUDA_RUNTIME(hipFree(a_h));
    CUDA_RUNTIME(hipFree(b_h));
    CUDA_RUNTIME(hipFree(c_h));
    break;
  default:
    fprintf(stderr, "unexpected AllocationType\n");
    exit(1);
  }

  a_h = nullptr;
  b_h = nullptr;
  c_h = nullptr;

  // fprintf(stderr, "destroy event\n");
  CUDA_RUNTIME(hipEventDestroy(kernelStart));
  CUDA_RUNTIME(hipEventDestroy(kernelStop));
  CUDA_RUNTIME(hipEventDestroy(txStart));
  CUDA_RUNTIME(hipEventDestroy(txStop));
  CUDA_RUNTIME(hipEventDestroy(rxStart));
  CUDA_RUNTIME(hipEventDestroy(rxStop));

  double copyPerf =
      1000.0 * n * sizeof(T) * 3 / (txMillis + rxMillis) / 1024 / 1024;
  double kernelPerf = 1000.0 * n * sizeof(T) * 3 / kernelMillis / 1024 / 1024;
  double totalPerf = 1000.0 * n * sizeof(T) * 3 / totalMillis / 1024 / 1024;

  // no copies in some of these
  if (at == ZERO_COPY) {
    copyPerf = -1;
  }
  if ((at == MANAGED) && (hint == NONE)) {
    copyPerf = -1;
  }
  if ((at == SYSTEM)) {
    copyPerf = -1;
  }

  Result result;
  result.kernel = kernelPerf;
  result.copy = copyPerf;
  result.total = totalPerf;
  // printf("%f.2 %f.2 %f.2\n", copyPerf, kernelPerf, totalPerf);
  return result;
}

void print_results(const std::vector<Result> results, const std::string &sep) {

  for (auto &result : results) {
    printf("%s", sep.c_str());
    if (result.copy >= 0) {
      printf("%.2e", result.copy);
    }
  }
  for (auto &result : results) {
    printf("%s", sep.c_str());
    if (result.kernel >= 0) {
      printf("%.2e", result.kernel);
    }
  }
  for (auto &result : results) {
    printf("%s", sep.c_str());
    if (result.total >= 0) {
      printf("%.2e", result.total);
    }
  }
  std::cout << std::endl;
}

template <typename T> std::vector<Result> run_many(size_t iters, T fn) {
  std::vector<Result> results;
  for (size_t i = 0; i < iters; ++i) {
    auto result = fn();
    results.push_back(result);
  }
  return results;
}

int main(int argc, char **argv) {

  bool disableSystemAllocator;

  // test the system allocator in a new process
  pid_t pid = fork(); // create child process
  int status;
  switch (pid)
  {
  case -1: // error
      perror("fork");
      exit(1);

  case 0: // child process 
      if (try_system_allocator()) {
        exit(0);
      }
      exit(1);

  default: // parent process, pid now contains the child pid
      while (-1 == waitpid(pid, &status, 0)); // wait for child to complete
      if (WIFSIGNALED(status) || WEXITSTATUS(status) != 0)
      {
          fprintf(stderr,"system allocator did not work (%d). disabling\n", status);
          disableSystemAllocator = true;
      } else {
        disableSystemAllocator = false;
      }
      break;
  }


  CUDA_RUNTIME(hipDeviceReset());

  std::string sep = ",";
  size_t iters = 5;

  cxxopts::Options options("triad", "triad benchmarks");

  std::vector<double> gs;
  std::vector<double> ms;

  options.add_options()("n,num-iters", "Number of iterations",
                        cxxopts::value(iters))(
      "no-system-allocator", "Disable system allocator")("h,help", "Show help");

  auto result = options.parse(argc, argv);

  const bool help = result["help"].as<bool>();
  if (help) {
    printf("%s\n", options.help().c_str());
    exit(0);
  }

  if (result["no-system-allocator"].count()) {
    disableSystemAllocator = true;
  }


  // print header
  std::cout << "n" << sep << "bmark";
  for (size_t i = 0; i < iters; ++i) {
    std::cout << sep << "copy_" + std::to_string(i);
  }
  for (size_t i = 0; i < iters; ++i) {
    std::cout << sep << "kernel_" + std::to_string(i);
  }
  for (size_t i = 0; i < iters; ++i) {
    std::cout << sep << "total_" + std::to_string(i);
  }
  std::cout << std::endl;

  // runs
  // 3GB => 1GB each => n=250M
  for (size_t n = 1e5; n <= 2.5e8; n *= 1.3) {

    std::vector<Result> results;
    if (!disableSystemAllocator) {
      results =
          run_many(iters, std::bind(benchmark_naive<int>, n, SYSTEM, NONE));
      printf("%.2e%s%s", (double)n, sep.c_str(), "system          ");
      print_results(results, sep);
    }

    results =
        run_many(iters, std::bind(benchmark_naive<int>, n, PAGEABLE, NONE));
    printf("%.2e%s%s", (double)n, sep.c_str(), "pageable          ");
    print_results(results, sep);

    results = run_many(iters, std::bind(benchmark_naive<int>, n, PINNED, NONE));
    printf("%.2e%s%s", (double)n, sep.c_str(), "pinned            ");
    print_results(results, sep);

    results =
        run_many(iters, std::bind(benchmark_naive<int>, n, ZERO_COPY, NONE));
    printf("%.2e%s%s", (double)n, sep.c_str(), "zero-copy         ");
    print_results(results, sep);

    results =
        run_many(iters, std::bind(benchmark_naive<int>, n, MANAGED, NONE));
    printf("%.2e%s%s", (double)n, sep.c_str(), "um                ");
    print_results(results, sep);

    results =
        run_many(iters, std::bind(benchmark_naive<int>, n, MANAGED, ACCESS));
    printf("%.2e%s%s", (double)n, sep.c_str(), "um-access         ");
    print_results(results, sep);

    results =
        run_many(iters, std::bind(benchmark_naive<int>, n, MANAGED, PREFETCH));
    printf("%.2e%s%s", (double)n, sep.c_str(), "um-prefetch       ");
    print_results(results, sep);

    results = run_many(
        iters, std::bind(benchmark_naive<int>, n, MANAGED, ACCESS | PREFETCH));
    printf("%.2e%s%s", (double)n, sep.c_str(), "um-access-prefetch");
    print_results(results, sep);
  }
}